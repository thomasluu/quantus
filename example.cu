#include "hip/hip_runtime.h"
#include <float.h>
#include <stdio.h>

#include <hiprand.h>

#include "include/quantus_gamma.h"

typedef double FP;
const FP alpha = 4;

__global__ void gamma_kernel(const FP *U, FP *X, unsigned int N, quantus_comm<FP> comm) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        X[i] = quantus_gamma(U[i], &comm);
    }
}

int main()
{
    const unsigned nblocks = 19532;
    const unsigned nthreads = 512;
    const unsigned n = nblocks * nthreads;
    const size_t size = n * sizeof(FP);

    FP *d_U, *d_X;
    hipMalloc((void **) &d_U, size);
    hipMalloc((void **) &d_X, size);

    FP *h_U = (FP *) malloc(size);
    FP *h_X = (FP *) malloc(size);

    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    if (sizeof(FP) == sizeof(float)) {
        hiprandGenerateUniform(gen, (float *) d_U, n);
    } else {
        hiprandGenerateUniformDouble(gen, (double *) d_U, n);
    }

    quantus_comm<FP> comm;
    quantus_gamma_cuda_init(alpha, &comm);

    gamma_kernel<<<nblocks, nthreads>>>(d_U, d_X, n, comm);

    quantus_cuda_cleanup(&comm);

    hipMemcpy(h_X, d_X, n * sizeof(FP), hipMemcpyDeviceToHost);
    hipMemcpy(h_U, d_U, n * sizeof(FP), hipMemcpyDeviceToHost);

    int digits = sizeof(FP) == sizeof(double) ? DBL_DIG : FLT_DIG;
    for (int i = 0; i < 10; i++) {
        printf("%.*e\t%.*e\n", digits, h_U[i], digits, h_X[i]);
    }

    free(h_U);
    free(h_X);

    hipFree(d_U);
    hipFree(d_X);

    hiprandDestroyGenerator(gen);
}
